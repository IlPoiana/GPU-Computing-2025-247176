#include "hip/hip_runtime.h"
//COO for computing on random matrices

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Device code
__global__ void COO_k(float* value, float* row, float* col, int res)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < N)
        {if(arr[col[i]] == 0)
            continue;
        res[row[i]] += value[i] * arr[col[i]];} 
}

__global__ resoid COO_1(float* value, float* row, float* col, int res)
{
    
    
}

__global__ void COO_Unrolled(float* value, float* row, float* col, int res)
{
    
}