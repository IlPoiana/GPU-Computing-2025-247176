#include "hip/hip_runtime.h"
//COO for computing on random matrices

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

struct int_matrix {
    int x;
    int y;
    int n;
    int * row;
    int * col;
    int * val;
};

struct int_extended_mtx{
        struct int_matrix mtx;
        int * arr;
};

// Std. implementation -> coalesced access and atomic add

// Row. implementation


// Reduction implementation
__global__ void row_spmv(int * val, int * arr, int * res,int start , int res_idx){
    int threads_num = gridDim.x * blockDim.x;
    int local_tid = blockIdx.x * blockDim.x + threadIdx.x; 
    int tid = local_tid + start;
    arr[tid] *= val[tid];
    //here start reduction
    if(threads_num == 1){
        res[res_idx] = arr[tid];
        return;
    }
    __syncthreads();
    //check if it's uneven
    if((threads_num & 1) == 1)
        if(tid == start)
            arr[tid] += arr[tid + threads_num - 1];

    for(int s = (threads_num/2); s > 0; s >>= 1 ){
        __syncthreads();
        if(s == 1 && local_tid == 0)
            res[res_idx] = arr[tid] + arr[tid + s];//saving one access to memory
        else if(local_tid < s)
            arr[tid] += arr[tid + s];
        
    }
    
    // res[res_idx] = arr[tid]; with shared is better this
}

// Std implementation + vector data structure + reduction

__global__ void COO_Unrolled(float* value, float* row, float* col, int res)
{
    
}

__global__ void COO_1(float* value, float* row, float* col, int res)
{
    
    
}
// Good idea but to develop further, further problems comes with the blocks that have to synchronize 
void host_row_spmv(int * d_val, int * d_arr, int * d_res,int * row, int n){
    int thread_n = 1; //number of threads to assign to each kernel
    int block_n = 1;
    for(int i = 1; i < n; i++){
        if(row[i] != row[i - 1]){
            printf("i: %d - row[i]: %d - thread_n: %d\n",i,row[i], thread_n);
            block_n = (thread_n / 256);
            if( block_n > 0){
                row_spmv<<<block_n,256>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
                thread_n -= block_n * 256;
            }
            if(thread_n > 0)
                row_spmv<<<1,thread_n>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
            thread_n = 1;//reset thread number
        }
        else{
            ++thread_n;
        }
    }
    //launch the last kernel
    printf("n: %d - row[n-1]: %d - thread_n: %d\n",n,row[n-1], thread_n);
    block_n += (thread_n / 256);
    if( block_n > 0){
        row_spmv<<<block_n,256>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
        thread_n -= block_n * 256;
    }
    if(thread_n > 0)
        row_spmv<<<1,thread_n>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
}

struct int_matrix gen_rnd_COO(int x, int y, int p, int binary){
    if(binary<1){
        printf("passed an invalid binary argument\n");
        struct int_matrix err = {0,0,0,0,0,0};
        return err;
    }

    int max_v = x * y;
    int * val = (int*)malloc(sizeof(int) * max_v);
    int * row = (int*)malloc(sizeof(int) * max_v);
    int * col = (int*)malloc(sizeof(int) * max_v);
    
    int count = 0;
    int v_buff = 0;

    for(int i = 0; i < max_v;i++){
        v_buff = rand()%p;
        if(v_buff == 0){
            if(binary == 1){
                val[count] = 1;
            }
            else{
                val[count] = (rand() % binary) + 1; 
            }
            row[count] = i / y; 
            col[count] = abs(y - i) % y;
            count = count + 1;
        }    
    }
    int * fval = (int*)malloc(sizeof(int) * count);
    int * frow = (int*)malloc(sizeof(int) * count);
    int * fcol = (int*)malloc(sizeof(int) * count);
    for( int i = 0; i< count; i++){
        fval[i] = val[i];
        frow[i] = row[i];
        fcol[i] = col[i];
    }

    struct int_matrix mtx = {x,y,count,frow,fcol,fval};
    return mtx;
}

int main(int argc, char *args[]){
    if (argc < 2 | argc < 3)
    {
        printf("require row and col number\n");
        return 0;
    }
    if (argc < 4){
        printf("insert the x: 1/ x probabilty to generate a value\n");
        return 0;
    }
    if (argc < 5){
        printf("tell if the random COO should be binary or not (>1)");
        return 0;
    }
    if (argc < 6){
        printf("tell the number of WARM-UP rounds");
        return 0;
    }
    if (argc < 7){
        printf("tell the number of ITERATION rounds");
        return 0;
    }
    //random initializer
    srand(time(NULL));


    int row_n = atoi(args[1]);
    int col_n = atoi(args[2]); 
    int max_v = row_n * col_n;
    int p = atoi(args[3]);
    int binary = atoi(args[4]);
    int warm_up = atoi(args[5]);
    int iterations = atoi(args[6]);
    // printf("Passed arguments\nx: %d\ny: %d\np: %d\nbinary: %d\n\n",row_n,col_n,p,binary);
   
    
    struct int_matrix mtx = gen_rnd_COO(row_n, col_n,p,binary);
    // PRINT_INT_MTX(mtx, COO);
    // printf("Running sparse matrix multiplication between a 1 vector and a integer value matrix\n");
    if (mtx.x == 0 && mtx.y == 0){
        printf("matrix not loaded correctly\n");
        return 0;
    }
    int n = mtx.n;
    int *row = mtx.row;
    int *col = mtx.col;
    int *value = mtx.val;
    int *res = (int*)calloc(row_n, sizeof(int));
    int * arr = (int*)malloc(sizeof(int) * mtx.y);
    double * measures = (double*)malloc(sizeof(double) * iterations);

    for(int i = 0; i<mtx.y; i++){
        arr[i] = 1;
        // arr[i] = rand() % 1000000;//for not have to handle out of memory cases
    }

    
    int * full_arr = (int*)malloc(sizeof(int) * mtx.n);
    for(int elem = 0; elem< mtx.n; elem++)
        full_arr[elem] = arr[mtx.col[elem]];       
        
    
    int *d_val, *d_arr, *d_res;
    hipMalloc(&d_val, n * sizeof(int));
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_res, mtx.y * sizeof(int));

    // Copy data to device
    hipMemcpy(d_val, value, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr, full_arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, mtx.y * sizeof(int), hipMemcpyHostToDevice);
    
    host_row_spmv(d_val,d_arr,d_res,row,n);

    // Copy results back to host
    hipMemcpy(res, d_res, mtx.y * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Result arr: ");
    for (int i = 0; i < mtx.y; i++) {
        printf("%d ", res[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_val);
    hipFree(d_arr);
    hipFree(d_res);
    return 0;
}