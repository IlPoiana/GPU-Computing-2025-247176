//COO for computing on random matrices


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

struct int_matrix {
    int x;
    int y;
    int n;
    int * row;
    int * col;
    int * val;
};

void reset_array(int * arr, int dim){
    for(int i=0; i<dim;i++){
        arr[i] = 0;
    }
}

float avg (float * arr, int nelem){
    float average = 0;
    // printf("nelem: %d\n", nelem);
    for(int i = 0; i< nelem; i++){
        average = average + arr[i] / nelem;
    }
    return average;
}

float std_dev(const float *data, float avg,int n) {
    float sum_sq = 0.0;
    for (int i = 0; i < n; i++) {
        float d = data[i] - avg;
        sum_sq += d * d;
    }
    
    return sqrt(
        sum_sq / n
    );
}

void convert_to_microseconds(float * measures, int dim){
    for(int i = 0; i < dim; i++){
        measures[i] *= 1000.0f;
    }
}

void JSON_FORMAT_ITER(int warm, int iter, float avg, float std){
    printf("{\"warmup\":%d,\"niter\":%d,\"avg\": %f,\"std\": %f},\n",warm,iter,avg,std);
}

// Std. implementation -> atomic add
__global__ void spmv_coo(int *val, int *row, int *col, int *arr, int *res, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Thread index
    if (tid < n) { 
        int row_idx = row[tid];  // Row index of the current non-zero element
        int vector_v = arr[col[tid]];  // Column index of the current non-zero element
        int product = val[tid] * vector_v;

        // Perform SpMV operation for this element
        atomicAdd(&res[row_idx], product);  
    }
}

// Std. implementation -> coalesced access and atomic add
__global__ void spmv_coo_coalesced(int *val, int *row, int * full_arr, int *res, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Thread index
    if (tid < n) { 
        int row_idx = row[tid];  
        int product = val[tid] * full_arr[tid];   

        // Perform SpMV operation for this element
        atomicAdd(&res[row_idx], product); 
    }
}

// Row. implementation -> one kernel with one block for each row: < thread num. case //TO DO
// __global__ void spmv_coo_block_reduction(int *val, int row_idx, int * full_arr, int *res, int start) {
//     int tid = threadIdx.x + start;  // Thread index 0..thread_n  
//     full_arr[tid] += val[tid] * full_arr[tid];  
//     __syncthreads(); 
//     parallel_reduction()
//     res[row_idx] = full_arr //store to the result, only thread of block 0 
    
// }

// Row. implementation -> one kernel with one block for each row: < thread num. case
// __global__ void spmv_coo_big_block_reduction(int *val, int row_idx, int * full_arr, int *res, int start, int row_length) {
//     int computation_n;
//     if(row_length % blockDim.x == 0)
//         computation_n = row_length / blockDim.x;
//     else
//         computation_n = row_length / blockDim.x + 1; // 11/10 = 2, 19/10 = 2, 20/10 = 2, 23/10 = 3;
//     int active_threads_n = row_length / computation_n;
//     int remaining_computation = row_length % (active_threads_n * computation_n); 
    
//     //11 10-> computation_n = 2 act = 5 rem = 11 % 10 = 1
//     // 19 10 -> computation_n = 2 act = 9 rem = 19 % 18 = 1
//     // 20 10 -> computation_n = 2 act = 10 rem 20%20 = 0
//     //134 -> comp_n = 14 act = 9 rem = 134 % 126 = 8

//     int local_idx = threadIdx.x * computation_n;
//     int tid = local_idx + start;

//     if(threadIdx.x > active_threads_n){
//         return; //don't compute 
//     }
    
//     int f_value = val[tid] * full_arr[tid]; //0 -> 0; 1 -> 2; 2 -> 4
    
//     if(threadIdx.x < active_threads_n)
//         for(int s = 1; s < computation_n; s++)
//                 f_value += val[tid + s] * full_arr[tid + s];
//     else
//         for(int s = 1; s < remaining_computation; s++)
//             f_value += val[tid + s] * full_arr[tid + s];
//     full_arr[tid] = f_value;
        
//     __syncthreads()
//     //now i have less than num threads reduction to do
//     //i have to move by computation_n
//     // for (int stride = (computation_n* active_threads_n / 2); stride > 0; stride >>= 1) {
//     //     __syncthreads();

//     //     if (local_idx < stride && (local_idx + stride + computation_n) < row_length) {
//     //         data[tid] += data[tid + stride + computation_n];
//     //     }
//     // }
    
//     if(threadIdx.x == 0){
//         for(int offset = computation_n; offset< row_length; offset += offset)//iterate over all the others values
//             f_value += full_arr[tid + offset]; 
//         res[tid] = f_value;//update res
//     }
        
// }

// Good idea but to develop further, further problems comes with the blocks that have to synchronize 
// void host_row_spmv(int * d_val, int * d_arr, int * d_res,int * row, int n){
//     int thread_n = 1; //number of threads to assign to each kernel
//     int block_n = 1;
//     for(int i = 1; i < n; i++){
//         if(row[i] != row[i - 1]){
//             printf("i: %d - row[i]: %d - thread_n: %d\n",i,row[i], thread_n);
//             block_n = (thread_n / 256);
//             if( block_n > 0){
//                 row_spmv<<<block_n,256>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
//                 thread_n -= block_n * 256;
//             }
//             if(thread_n > 0)
//                 row_spmv<<<1,thread_n>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
//             thread_n = 1;//reset thread number
//         }
//         else{
//             ++thread_n;
//         }
//     }
//     //launch the last kernel
//     printf("n: %d - row[n-1]: %d - thread_n: %d\n",n,row[n-1], thread_n);
//     block_n += (thread_n / 256);
//     if( block_n > 0){
//         row_spmv<<<block_n,256>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
//         thread_n -= block_n * 256;
//     }
//     if(thread_n > 0)
//         row_spmv<<<1,thread_n>>>(d_val,d_arr,d_res,i - thread_n,row[i - 1]);
// }

struct int_matrix gen_rnd_COO(int x, int y, int p, int binary){
    if(binary<1){
        printf("passed an invalid binary argument\n");
        struct int_matrix err = {0,0,0,0,0,0};
        return err;
    }

    long max_v = x * y;
    int * val = (int*)malloc(sizeof(int) * max_v);
    int * row = (int*)malloc(sizeof(int) * max_v);
    int * col = (int*)malloc(sizeof(int) * max_v);
    
    int count = 0;
    int v_buff = 0;

    for(int i = 0; i < max_v;i++){
        v_buff = rand()%p;
        if(v_buff == 0){
            if(binary == 1){
                val[count] = 1;
            }
            else{
                val[count] = (rand() % binary) + 1; 
            }
            row[count] = i / y; 
            col[count] = abs(y - i) % y;
            count = count + 1;
        }    
    }
    int * fval = (int*)malloc(sizeof(int) * count);
    int * frow = (int*)malloc(sizeof(int) * count);
    int * fcol = (int*)malloc(sizeof(int) * count);
    for( int i = 0; i< count; i++){
        fval[i] = val[i];
        frow[i] = row[i];
        fcol[i] = col[i];
    }

    struct int_matrix mtx = {x,y,count,frow,fcol,fval};
    return mtx;
}

int main(int argc, char *args[]){
    if (argc < 2 | argc < 3)
    {
        printf("require row and col number\n");
        return 0;
    }
    if (argc < 4){
        printf("insert the x: 1/ x probabilty to generate a value\n");
        return 0;
    }
    if (argc < 5){
        printf("tell if the random COO should be binary or not (>1)");
        return 0;
    }
    if (argc < 6){
        printf("tell the number of WARM-UP rounds");
        return 0;
    }
    if (argc < 7){
        printf("tell the number of ITERATION rounds");
        return 0;
    }
    if (argc < 8){
        printf("tell the number of threads");
        return 0;
    }
    //random initializer
    srand(time(NULL));


    int row_n = atoi(args[1]);
    int col_n = atoi(args[2]); 
    int max_v = row_n * col_n;
    int p = atoi(args[3]);
    int binary = atoi(args[4]);
    int warm_up = atoi(args[5]);
    int iterations = atoi(args[6]);
    int thread_n = atoi(args[7]);
    // printf("Passed arguments\nx: %d\ny: %d\np: %d\nbinary: %d\n\n",row_n,col_n,p,binary);
   
    
    struct int_matrix mtx = gen_rnd_COO(row_n, col_n,p,binary);
    // PRINT_INT_MTX(mtx, COO);
    // printf("Running sparse matrix multiplication between a 1 vector and a integer value matrix\n");
    if (mtx.x == 0 && mtx.y == 0){
        printf("matrix not loaded correctly\n");
        return 0;
    }
    int n = mtx.n;
    int *row = mtx.row;
    int *col = mtx.col;
    int *value = mtx.val;
    int *res = (int*)calloc(row_n, sizeof(int));
    int * arr = (int*)malloc(sizeof(int) * mtx.y);
    float * measures = (float*)malloc(sizeof(float) * iterations);
    float average = 0;

    for(int i = 0; i<mtx.y; i++){
        // arr[i] = 1;
        arr[i] = rand() % 1000000;//for not have to handle out of memory cases
    }

    
    int * full_arr = (int*)malloc(sizeof(int) * mtx.n);
    for(int elem = 0; elem< mtx.n; elem++)
        full_arr[elem] = arr[mtx.col[elem]];       
        
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *d_val, * d_row,*d_col,*d_short_arr,*d_arr, *d_res;
    hipMalloc(&d_val, n * sizeof(int));
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_short_arr, mtx.y * sizeof(int));
    hipMalloc(&d_col, n * sizeof(int));
    hipMalloc(&d_row, n * sizeof(int));
    hipMalloc(&d_res, mtx.x * sizeof(int));

    // Copy data to device
    hipMemcpy(d_val, value, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_short_arr, arr, mtx.y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr, full_arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, row, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, col, n * sizeof(int), hipMemcpyHostToDevice);
    
    int block_n = (n / thread_n) + 1;

    //initialize json
    printf("{\"x\":%d,\n\"y\":%d,\"n\":%d,\"threads_n\":%d,\"runs\":{",mtx.x,mtx.y,mtx.n,thread_n);

    for(int i = -warm_up; i< iterations; i++){
        if(i < 0 ){
            spmv_coo<<<block_n,thread_n>>>(d_val, d_row, d_col,d_short_arr, d_res,n);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);//zeroing the memory      
        }
        else{
            hipEventRecord(start);
            spmv_coo<<<block_n,thread_n>>>(d_val, d_row, d_col,d_short_arr, d_res,n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&measures[i], start, stop);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
        }
    }
    convert_to_microseconds(measures, iterations);
    average = avg(measures, iterations);
    printf("\"std.\":");
    JSON_FORMAT_ITER(warm_up,iterations,average,std_dev(measures,average,iterations));
    
    // printf("Result arr: ");
    // for (int i = 0; i < mtx.x; i++) {
    //     printf("%d ", res[i]);
    // }
    // printf("\n");

    for(int i = -warm_up; i< iterations; i++){
        if(i < 0 ){
            spmv_coo_coalesced<<<block_n,thread_n>>>(d_val, d_row,d_arr, d_res,n);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);//zeroing the memory      
        }
        else{
            hipEventRecord(start);
            spmv_coo_coalesced<<<block_n,thread_n>>>(d_val, d_row,d_arr, d_res,n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&measures[i], start, stop);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
        }
    }
    convert_to_microseconds(measures, iterations);
    average = avg(measures, iterations);
    printf("\"std. coalesced\":");
    printf("{\"warmup\":%d,\"niter\":%d,\"avg\": %f,\"std\": %f}\n",warm_up,iterations,average,std_dev(measures,average,iterations));
    
    printf("}}\n");
    // Free device memory
    hipFree(d_val);
    hipFree(d_short_arr);
    hipFree(d_arr);
    hipFree(d_res);
    hipFree(d_row);
    hipFree(d_col);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}