// COO for running imported matrices


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

struct matrix{
    int x;
    int y;
    int n;
    int * row;
    int * col;
};

struct int_matrix {
    int x;
    int y;
    int n;
    int * row;
    int * col;
    int * val;
};

float avg (float * arr, int nelem){
    float average = 0;
    // printf("nelem: %d\n", nelem);
    for(int i = 0; i< nelem; i++){
        average = average + arr[i] / nelem;
    }
    return average;
}

float std_dev(const float *data, float avg,int n) {
    float sum_sq = 0.0;
    for (int i = 0; i < n; i++) {
        float d = data[i] - avg;
        sum_sq += d * d;
    }
    
    return sqrt(
        sum_sq / n
    );
}

void convert_to_microseconds(float * measures, int dim){
    for(int i = 0; i < dim; i++){
        measures[i] *= 1000.0f;
    }
}

void JSON_FORMAT_ITER(int warm, int iter, float avg, float std){
    printf("{\"warmup\":%d,\"niter\":%d,\"avg\": %f,\"std\": %f},\n",warm,iter,avg,std);
}

struct matrix import_matrix(char * file_path){
    FILE *file = fopen(file_path, "r");

    // Check if the file was opened successfully
    if (file == NULL) {
        perror("Error opening file");
        struct matrix err = {0,0,0,0,0};
        return err;
    }

    char myString[128] = "";
    int init = 1; //flag for initialize the number of elements
    
    int x; int y; int n;
    
    int *row; int *col;

    int index = 0;
    // Read the content and print it
    while(fgets(myString, 100, file)) {
        
        if(myString[0] == (char)37){}
        else if (init == 1){
            char dimx[16] = ""; char dimy[16] = "";char nelem[16] = "";
            int count = 3;
            for(int i = 0; (int)myString[i] != '\n'; i++){
                if((int)myString[i] != ' '){
                    char temp[2] = {myString[i],'\0'};
                    switch (count)
                    {
                    case 3:
                        strcat(dimx, temp);
                        break;
                    case 2:
                        strcat(dimy, temp); 
                        break;
                    case 1:
                        strcat(nelem, temp);
                        break;
                    default:
                        printf("why are you here!\n");
                        break;
                    }
                }
                else{
                    count = count - 1;
                }
                 
            }
                init = 0;
                x = atoi(dimx);y = atoi(dimy);n = atoi(nelem);
                fflush(stdout);
                row = (int*)malloc(sizeof(int) * n);
                col = (int*)malloc(sizeof(int) * n);
        }
        else{
            int count = 2;
            char xelem[16] = "";char yelem[16] = "";char c;  
            for(int i = 0; (int)myString[i] != '\n'; i++){
                if((int)myString[i] != ' '){
                    char temp[2] = {myString[i],'\0'};
                    switch (count)
                    {
                    case 2:
                        strcat(xelem, temp);
                        break;
                    case 1:
                        strcat(yelem, temp);
                        break;
                    default:
                        printf("why are you here!\n");
                        break;
                    }
                }
                else{
                    count = count - 1;
                }
            }
            row[index] = atoi(xelem);col[index] = atoi(yelem);
            index = index + 1;
        }
    }

    // Close the file after finishing
    fclose(file);  
    //switch col and row
    struct matrix sm = {x,y,n,col, row};
    return sm;  
}

struct int_matrix import_int_matrix(char *file_path) {
    FILE *file = fopen(file_path, "r");

    if (file == NULL) {
        perror("Error opening file");
        struct int_matrix err = {0, 0, 0, NULL, NULL, NULL};
        return err;
    }

    char line[128];
    int x = 0, y = 0, n = 0;
    int *row = NULL, *col = NULL, *val = NULL;

    // Read until you get dimensions
    while (fgets(line, sizeof(line), file)) {
        if (line[0] == '%') continue; // comment line
        if (sscanf(line, "%d %d %d", &x, &y, &n) == 3) {
            // allocate memory now
            row = (int*)malloc(n * sizeof(int));
            col = (int*)malloc(n * sizeof(int));
            val = (int*)malloc(n * sizeof(int));
            if (!row || !col || !val) {
                perror("Memory allocation failed");
                fclose(file);
                struct int_matrix err = {0, 0, 0, NULL, NULL, NULL};
                return err;
            }
            break;
        } else {
            fprintf(stderr, "Invalid matrix dimension line format\n");
            fclose(file);
            struct int_matrix err = {0, 0, 0, NULL, NULL, NULL};
            return err;
        }
    }

    int index = 0;
    while (index < n && fgets(line, sizeof(line), file)) {
        if (line[0] == '%') continue; // skip comments

        int r, c, v;
        if (sscanf(line, "%d %d %d", &r, &c, &v) == 3) {
            row[index] = r;
            col[index] = c;
            val[index] = v;
            index++;
        } else {
            fprintf(stderr, "Invalid matrix element line format at index %d\n", index);
            // Handle error or skip line, here we abort:
            free(row);
            free(col);
            free(val);
            fclose(file);
            struct int_matrix err = {0, 0, 0, NULL, NULL, NULL};
            return err;
        }
    }

    fclose(file);

    struct int_matrix sm = {x, y, n, row, col,val}; // note: original swaps col and row here, preserve that if needed
    return sm;
}

struct int_matrix convert_to_int_mtx(struct matrix tmp){
    struct int_matrix mtx = {tmp.x,tmp.y,tmp.n,tmp.row,tmp.col,NULL};
    int * val = (int*)malloc(tmp.n * sizeof(int));
    for(int i = 0; i < tmp.n; i++){
        val[i] = 1;
    }
    mtx.val = val;
    return mtx;
}

// Std. implementation -> atomic add
__global__ void spmv_coo(int *val, int *row, int *col, int *arr, int *res, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Thread index
    if (tid < n) { 
        int row_idx = row[tid];  // Row index of the current non-zero element
        int vector_v = arr[col[tid]];  // Column index of the current non-zero element
        int product = val[tid] * vector_v;

        // Perform SpMV operation for this element
        atomicAdd(&res[row_idx], product);  
    }
}

// Std. implementation -> coalesced access and atomic add
__global__ void spmv_coo_coalesced(int *val, int *row, int * full_arr, int *res, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Thread index
    if (tid < n) { 
        int row_idx = row[tid];  
        int product = val[tid] * full_arr[tid];   

        // Perform SpMV operation for this element
        atomicAdd(&res[row_idx], product); 
    }
}



int main(int argc, char *args[]){
    if (argc < 2)
    {
        printf("require filepath!");
        return 0;
    }
    if (argc < 3)
    {
        printf("insert 0 binary or 1 integer mtx\n");
        return 0;
    }
    if (argc < 4)
    {
        printf("insert 1 if you want all 1 vector, or the 1/p probabilty to generate a value\n");
        return 0;
    }
    if (argc < 5)
    {
        printf("tell the number of WARM-UP rounds");
        return 0;
    }
    if (argc < 6)
    {
        printf("tell the number of ITERATION rounds");
        return 0;
    }
    if (argc < 7)
    {
        printf("tell the number of threads per block");
        return 0;
    }
    //random initializer
    srand(time(NULL));


    char *file_path = args[1];
    int type = atoi(args[2]);
    int p = atoi(args[3]);
    int warm_up = atoi(args[4]);
    int iterations = atoi(args[5]);
    int thread_n = atoi(args[6]);

    struct int_matrix mtx;
    if (type == 0){
        struct matrix tmp = import_matrix(file_path);
        mtx = convert_to_int_mtx(tmp); 
    }
    else if (type == 1)
        mtx = import_int_matrix(file_path); 
    else{
        printf("wrong import format!\n");
        return 0;
    }

    int n = mtx.n;
    int *row = mtx.row;
    int *col = mtx.col;
    int *value = mtx.val;
    int *res = (int*)calloc(mtx.x, sizeof(int));
    int * arr = (int*)malloc(sizeof(int) * mtx.y);
    float * measures = (float*)malloc(sizeof(float) * iterations);
    float average = 0;

    for(int i = 0; i<mtx.y; i++){
        // arr[i] = 1;
        arr[i] = rand() % 1000000;//for not have to handle out of memory cases
    }

    
    int * full_arr = (int*)malloc(sizeof(int) * mtx.n);
    for(int elem = 0; elem< mtx.n; elem++)
        full_arr[elem] = arr[mtx.col[elem]];       
        
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *d_val, * d_row,*d_col,*d_short_arr,*d_arr, *d_res;
    hipMalloc(&d_val, n * sizeof(int));
    hipMalloc(&d_arr, n * sizeof(int));
    hipMalloc(&d_short_arr, mtx.y * sizeof(int));
    hipMalloc(&d_col, n * sizeof(int));
    hipMalloc(&d_row, n * sizeof(int));
    hipMalloc(&d_res, mtx.x * sizeof(int));

    // Copy data to device
    hipMemcpy(d_val, value, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_short_arr, arr, mtx.y * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_arr, full_arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, row, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col, col, n * sizeof(int), hipMemcpyHostToDevice);
    
    int block_n = (n / thread_n) + 1;

    printf("non 0 elem: %d\n", mtx.n);
    //initialize json
    printf("{\"x\":%d,\n\"y\":%d,\"n\":%d,\"runs\":{",mtx.x,mtx.y,mtx.n);

    for(int i = -warm_up; i< iterations; i++){
        if(i < 0 ){
            spmv_coo<<<block_n,thread_n>>>(d_val, d_row, d_col,d_short_arr, d_res,n);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);//zeroing the memory      
        }
        else{
            hipEventRecord(start);
            spmv_coo<<<block_n,thread_n>>>(d_val, d_row, d_col,d_short_arr, d_res,n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&measures[i], start, stop);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
        }
    }
    convert_to_microseconds(measures, iterations);
    average = avg(measures, iterations);
    printf("\"std.\":");
    JSON_FORMAT_ITER(warm_up,iterations,average,std_dev(measures,average,iterations));
    
    // printf("Result arr: ");
    // for (int i = 0; i < mtx.x; i++) {
    //     printf("%d ", res[i]);
    // }
    // printf("\n");

    for(int i = -warm_up; i< iterations; i++){
        if(i < 0 ){
            spmv_coo_coalesced<<<block_n,thread_n>>>(d_val, d_row,d_arr, d_res,n);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);//zeroing the memory      
        }
        else{
            hipEventRecord(start);
            spmv_coo_coalesced<<<block_n,thread_n>>>(d_val, d_row,d_arr, d_res,n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&measures[i], start, stop);
            hipMemcpy(d_res, res, mtx.x * sizeof(int), hipMemcpyHostToDevice);
        }
    }
    convert_to_microseconds(measures, iterations);
    average = avg(measures, iterations);
    printf("\"std. coalesced\":");
    printf("{\"warmup\":%d,\"niter\":%d,\"avg\": %f,\"std\": %f}\n",warm_up,iterations,average,std_dev(measures,average,iterations));
    
    printf("}}\n");
    // Free device memory
    hipFree(d_val);
    hipFree(d_short_arr);
    hipFree(d_arr);
    hipFree(d_res);
    hipFree(d_row);
    hipFree(d_col);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}