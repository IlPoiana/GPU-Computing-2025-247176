
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void myKernel(int * x){
    *x = *x + 1; 
}

// KEY TAKEAWAY
/**
 * cudaMallocManaged() creates a "virtual shared memory" between GPU and CPU.
 * BUT!!! it overwrite the modifications to the variables in the host after the cudaMallocManaged call.
 */

int main(){
    printf("starting host execution\n");
    // int * x = (int*)malloc(sizeof(int)); // we don't need this
    int * x;
    // *x= 2; 
    
    //here, the global GPU memory doesn't see the modification done before this point.
    hipMallocManaged(&x, sizeof(int));
    printf("x: %d --> 0\n", *x);
    
    //now it see the "shared" adress of memory
    *x = 2;
    printf("x: %d --> 2\n", *x);
    myKernel<<<1,1>>>(x);
    hipDeviceSynchronize();

    printf("x: %d", *x);

    return 0;
}