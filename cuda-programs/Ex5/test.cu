
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

void print_arr(int * arr, int n){
    for(int i = 0; i < n; i++){
        printf(" %d ", arr[i]);
    }
}

// Host code
int main(int argc, char * args[])
{
    int N = atoi(args[1]);
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize input vectors
    srand(time(NULL));
    int * x1 = (int *)malloc(sizeof(int) * N);
    int * x2 = (int *)malloc(sizeof(int) * N);
    int * res = (int *)malloc(sizeof(int) * N);
    
    for(int i = 0; i<N; i++){
        x1[i] = rand() % 1 + 1;
        x2[i] = rand() % 1 + 1;
    }


    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    fflush(stdout);
    printf("res\n");
    print_arr(res,N);
    printf("\n");
    
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}