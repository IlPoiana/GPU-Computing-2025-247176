
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void add_kernel_naive(int * x1, int * x2, int * res, int n){
    for(int i = 0; i< n; i++){
        res[i] = x1[i] + x2[i];
    }
}

__global__ void add_kernel(int * x1, int * x2, int * res, int n){
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // if (n % ((blockIdx.x + 1) * blockDim.x) != 0){//means that is the block with more cores than threads
    //     if (index < (n % blockDim.x))
    //     {
    //         res[index] = x1[index] + x2[index];
    //     }    
    // }
    // else{
    //     res[index] = x1[index] + x2[index];

    // }
    res[index] = x1[index] + x2[index];
    
}

__global__ void add_kernel_jump(int * x1, int * x2, int * res, int n){
    for(int i = 0; i< n; i++){
        res[i] = x1[i] + x2[i];
    }
}

__global__ void plus_one(int * x1, int n){
    for(int i = 0; i< n; i++){
        x1[i]++;
    }
}

void print_arr(int * arr, int n){
    for(int i = 0; i < n; i++){
        printf(" %d ", arr[i]);
    }
}

int main(int argc, char * args[]){
    printf("starting host execution\n");
    if(argc < 1){
        printf("no dimension passed\n");
        return 0;
    }
    int N = atoi(args[1]);

    srand(time(NULL));
    int * x1;
    int * x2;
    int * res;
    
    int blockSize = 256;
    int blockNumber = (N + blockSize -1) / blockSize;
    

    // Create the memory in the GPU
    hipMallocManaged(&x1, sizeof(int) * N);
    hipMallocManaged(&x2, sizeof(int) * N);
    hipMallocManaged(&res, sizeof(int) * N);

    for(int i = 0; i<N; i++){
        x1[i] = rand() % 100;
        x2[i] = rand() % 100;
    }

    // printf("x1\n");
    // print_arr(x1, N);

    // printf("\nx2\n");
    // print_arr(x2, N);

    //Cuda event for kernel performance time measure
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, hipEventBlockingSync);
    hipEventCreateWithFlags(&stop, hipEventBlockingSync);
    
    hipEventRecord(start);
    //code here
    add_kernel<<<blockNumber,blockSize>>>(x1,x2,res,N);
    // plus_one<<<blockNumber,threadNumber>>>(x1,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop); //float
    
    float milliseconds = 0;
    
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nKernel Time: %f ms\n", milliseconds);
    
    // printf("res\n");
    // print_arr(res,N);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(x1);
    hipFree(x2);
    hipFree(res);
    return 0;
}