
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#ifndef TYPE
#define TYPE double
#endif

//Correct
__global__ void add_kernel(TYPE * x1, TYPE * x2, TYPE * res, int n){
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(index < n)
        res[index] = x1[index] + x2[index];
    
}

//Works only for `int` types
void print_arr(int * arr, int n){
    for(int i = 0; i < n; i++){
        printf(" %d ", arr[i]);
    }
}

int main(int argc, char * args[]){
    printf("starting host execution\n");
    if(argc < 1){
        printf("no dimension passed\n");
        return 0;
    }
    int N = atoi(args[1]);

    srand(time(NULL));
    TYPE * x1 = (TYPE *)malloc(sizeof(TYPE) * N);
    TYPE * x2 = (TYPE *)malloc(sizeof(TYPE) * N);
    TYPE * res = (TYPE *)malloc(sizeof(TYPE) * N);
    
    int blockSize = 256;
    int blockNumber = (N + blockSize -1) / blockSize;
    

    // Create the memory in the GPU
    hipMallocManaged(&x1, sizeof(TYPE) * N);
    hipMallocManaged(&x2, sizeof(TYPE) * N);
    hipMallocManaged(&res, sizeof(TYPE) * N);

    for(int i = 0; i<N; i++){
        x1[i] = rand() % 1 + 1;
        x2[i] = rand() % 1 + 1;
    }

    // printf("x1\n");
    // print_arr(x1, N);

    // printf("\nx2\n");
    // print_arr(x2, N);

    //Cuda event for kernel performance time measure
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, hipEventBlockingSync);
    hipEventCreateWithFlags(&stop, hipEventBlockingSync);
    
    hipEventRecord(start);
    //code here
    add_kernel<<<blockNumber,blockSize>>>(x1,x2,res,N);

    
    hipEventRecord(stop);
    hipEventSynchronize(stop); //float
    
    float milliseconds = 0;
    
    hipEventElapsedTime(&milliseconds, start, stop);
    

    // printf("res\n");
    // print_arr(res,N);

    if(sizeof(TYPE) == sizeof(int))
        printf("\niteration %d -- result checking: %d = 2\n", N, res[N - 1]);
    else if(sizeof(TYPE) == sizeof(float))
        printf("\niteration %d -- result checking: %f = 2\n", N, res[N - 1]);
    else if(sizeof(TYPE) == sizeof(double))
        printf("\niteration %d -- result checking: %2f = 2\n", N, res[N - 1]);
    printf("\nKernel Time: %f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(x1);
    hipFree(x2);
    hipFree(res);
    return 0;
}