#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void add_kernel(int * x1, int * x2, int * res, int n){
    for(int i = 0; i< n; i++){
        res[i] = x1[i] + x2[i];
    }
}

int main(){
    printf("starting host execution\n");

    int N = 5;
    srand(time(NULL));
    int * x1 = (int*)malloc(sizeof(int) * N);
    int * x2 = (int*)malloc(sizeof(int) * N);
    int * res = (int*)malloc(sizeof(int) * N);

    for(int i = 0; i<N; i++){
        x1[i] = rand() % 100;
        x2[i] = rand() % 100;
    }

    printf("x1\n");
    for(int i = 0; i < N; i++){
        printf(" %d ", x1[i]);
    }


    printf("x2\n");
    for(int i = 0; i < N; i++){
        printf(" %d ", x2[i]);
    }
    //Create the memory in the GPU
    hipMallocManaged(&x1, sizeof(int) * N);
    hipMallocManaged(&x2, sizeof(int) * N);
    hipMallocManaged(&res, sizeof(int) * N);

    //Cuda event for kernel performance time measure
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    //code here
    int blockNumber = 1;
    int threadNumber = 256;
    add_kernel<<<threadNumber,blockNumber>>>(x1,x2,res,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop); //float
    double milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Time: %2f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(x1);
    hipFree(x2);
    hipFree(res);
    return 0;
}